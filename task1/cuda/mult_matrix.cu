#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void matrixMult(const double *A, const double *B, double *C, int K, int N)
{
    int i0 = blockDim.y * blockIdx.y + threadIdx.y;
    int j0 = blockDim.x * blockIdx.x + threadIdx.x;
    
    double sum = 0;
    for (int k = 0; k < K; k++)
        sum += A[i0 * K + k] * B[k * N + j0];
    C[N * i0 + j0] = sum;
}

void init_matrix_rnd(double* &matrix, int number_row, int number_col)
{
	for (size_t i = 0; i < number_row * number_col; i++)
		matrix[i] = double(rand()) / double(1000);
}

int main()
{
    //start, stop - for Kernel time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // количество строк и столбцов матриц A[MxK] и B[KxN]
    int M = 32, K = 48, N = 32;
    // Размеры матриц A и B должны нацело делиться на размер блока.

    size_t Asize = M * K * sizeof(double);
    size_t Bsize = K * N * sizeof(double);
    size_t Csize = M * N * sizeof(double);

    double *h_A = (double *)malloc(Asize);
    double *h_B = (double *)malloc(Bsize);
    double *h_C = (double *)malloc(Csize);

    init_matrix_rnd(h_A, M, K);
    init_matrix_rnd(h_B, K, N);

    double *d_A = NULL;
    hipMalloc((void **)&d_A, Asize);

    double *d_B = NULL;
    hipMalloc((void **)&d_B, Bsize);

    double * d_C = NULL;
    hipMalloc((void **)&d_C, Csize);
    hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bsize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid = dim3(N / BLOCK_SIZE, M / BLOCK_SIZE);

    hipEventRecord(start, 0);

    matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, K, N);

    hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
    float KernelTime;
    hipEventElapsedTime( &KernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n", KernelTime);

    hipMemcpy(h_C, d_C, Csize, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy( start );
    hipEventDestroy( stop );

    return 0;
}